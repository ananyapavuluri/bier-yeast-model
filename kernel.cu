#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <iomanip>
#include <cstdlib>
#include <stdio.h>
#include <fstream>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime.h"
#include ""

using namespace std;

#define variations 2000

__global__ void simulation();
// we want all the params_all to be multiplied by everything in params_baseline

int main(void)
{

    double Vin = 0.36;
    double K1 = 0.02;
    double kp = 6;
    double Km = 13;

    double params_baseline = [Vin, k1, kp, Km];
    int n_params = 4;
    




}